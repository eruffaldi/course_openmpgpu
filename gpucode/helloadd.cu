	
#include <hip/hip_runtime.h>
#include <stdio.h>
	__global__ void add( int *a, int *b, int *c ) {
		*c = *a + *b;
	}
	int main( void ) {
		int a=2, b=7, c; // host copies of a, b, c
		int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
		int size = sizeof( int ); // we need space for an integer
		hipMalloc( (void**)&dev_a, size );
		hipMalloc( (void**)&dev_b, size );
		hipMalloc( (void**)&dev_c, size );
		hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice );
		hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice );
		add<<< 1, 1 >>>( dev_a, dev_b, dev_c );
		hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost );
		hipFree( dev_a );
		hipFree( dev_b );
		hipFree( dev_c );
		printf("Result %d\n",c);
		return 0;
	}