	
#include <hip/hip_runtime.h>
#include <stdio.h>
	__global__ void add( int *a, int *b, int *c ) {
		*c = *a + *b;
	}
	int main( void ) {
		int a=2, b=7, c; // host copies of a, b, c
		int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
		int size = sizeof( int ); // we need space for an integer
		hipMalloc( (void**)&dev_a, size );
		hipMalloc( (void**)&dev_b, size );
		hipMalloc( (void**)&dev_c, size );
		hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice );
		hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice );
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		
		add<<< 1, 1 >>>( dev_a, dev_b, dev_c );
		hipEventRecord(stop, 0);
hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost );
		hipEventSynchronize(stop);
		hipFree( dev_a );
		hipFree( dev_b );
		hipFree( dev_c );
		float time;
		hipEventElapsedTime(&time, start, stop);
		printf("Result %d Time %f \n",time,c);
		return 0;
	}