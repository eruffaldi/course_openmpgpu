
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


struct Timer
{

		hipEvent_t xstart, xstop;

		Timer()
		{
		hipEventCreate(&xstart);
		hipEventCreate(&xstop);
		}

		void start()
		{
		hipEventRecord(xstart, 0);

		}

		void stop()
		{
		hipEventRecord(xstop, 0);

		}

		void sync()
		{
		hipEventSynchronize(xstop);

		}

		float elapsed() const
		{
			float time = 0;
		hipEventElapsedTime(&time, xstart, xstop);
			return time;
		}
};

/*
A = Ar x Ac
B = Br x Bc
E faccio che Ac==Br

matrixmul<<dim3(Ar,Bc),Ac>>(a,b,c)
Rowmajor senza pitch
Assuming Ac < 1024
*/

template<int N,bool once>
__global__ void mmulk(float * a, float * b, float * c, int Br, int Bc)
{
	int ai,bi,ci; // usando threadIdx blockIdx blockDim
	__shared__ float tmp[N];

	ai = blockIdx.x* Br  + threadIdx.x; // a(blockIdx.x,threadIdx.x)
	bi = threadIdx.x*Bc + blockIdx.y;  // b(threadIdx.x,blokcIdx.y)

	tmp[threadIdx.x] = a[ai]*b[bi]; // along the vector
	__syncthreads();
	if(0 == threadIdx.x)
	{
		float x = 0;
		for(int i = 0; i < blockDim.x; i++)
			x += tmp[i];
		ci = blockIdx.x*Bc + blockIdx.y; //c(blockIdx.x,blockIdx.y)	
		if(once)
			c[ci] = x;
		else
			atomicAdd(c+ci,x); // atomic increment of the result in c[ci]
	}
}

template<class T,int base=0>
class MatrixWrap
{
public:
	MatrixWrap(T * p, int a,int b): pp(p),r(a),c(b) {}

	T operator() (int i,int j) const { return pp[(i-base)*c+(j-base)]; }

	T &operator() (int i,int j)  { return pp[(i-base)*c+(j-base)]; }

	int r,c;
	T * pp;
};

template <class T>
void mmul(MatrixWrap<T> a,MatrixWrap<T> b,MatrixWrap<T> c)
{
	// TODO assert
	for(int i = 0; i < c.r; i++)
		for(int j = 0; j < c.c; j++)
		{
			T t = 0;
			for(int k = 0; k < a.c; k++)
				t += a(i,k)*b(k,j);
			c(i,j) = t;
		}
}

template <class T>
void minit(MatrixWrap<T,0> a,T rv, T cv, T dv)
{
	for(int i = 0; i < a.r; i++)
		for(int j = 0; j < a.c; j++)
			a(i,j) = i*rv+j*cv+dv;

}

template <class T>
std::ostream & operator << (std::ostream & ons, MatrixWrap<T> & w)
{
	ons << "(" << w.r << "," << w.c << ")\n";
	for(int i = 0; i < w.r; i++)
	{
		for(int j = 0; j < w.c; j++)
			ons << w(i,j) << " ";
		ons << std::endl;
	}
	return ons;
}

int main(int argc, char const *argv[])
{
	int Ar = 1, Ac = 512, Bc = 1;
	int & Br = Ac;
	int sizeA = sizeof(float)*Ar*Ac;
	int sizeB = sizeof(float)*Ac*Bc;
	int sizeC = sizeof(float)*Ar*Bc;
	float *dev_a,*dev_b,*dev_c;
	std::vector<float> a(sizeA/sizeof(float));
	std::vector<float> b(sizeB/sizeof(float));
	std::vector<float> c(sizeC/sizeof(float));
	std::vector<float> cr(sizeC/sizeof(float));
	MatrixWrap<float,0> mwA(&a[0],Ar,Ac);
	MatrixWrap<float,0> mwB(&b[0],Ac,Bc);
	MatrixWrap<float,0> mwC(&c[0],Ar,Bc);
	MatrixWrap<float,0> mwCr(&cr[0],Ar,Bc);

	hipMalloc( (void**)&dev_a, sizeA );
	hipMalloc( (void**)&dev_b, sizeB );
	hipMalloc( (void**)&dev_c, sizeC);

	minit<float>(mwA,2,1,0);
	minit<float>(mwB,-8,-4,0);
	minit<float>(mwC,0,0,1);
	minit<float>(mwCr,0,0,2);

	mmul(mwA,mwB,mwC);

	std::cout << mwA << std::endl;
	std::cout << mwB << std::endl;
	std::cout << mwC << std::endl;
	Timer t;
	t.start();
	hipMemcpy(dev_a,mwA.pp,sizeA,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,mwB.pp,sizeB,hipMemcpyHostToDevice);
	if(Ac < 1024)
		mmulk<1024,true> <<<dim3(Ar,Bc),Ac>>>(dev_a,dev_b,dev_c,Br,Bc); // no atomic, just once
	else
	{
		//cudaMemset(dev_c,0,sizeC); // when using atomic
		//mmulk<1024,false> <<<dim3(Ar,Bc),512>>>(dev_a,dev_b,dev_c,Br,Bc);
		std::cout << "NOT IMPLEMENTED\n";
	}
	hipMemcpy(mwCr.pp,dev_c,sizeC,hipMemcpyDeviceToHost);
	t.stop();
	t.sync();
	std::cout << mwCr << std::endl;
	std::cout << t.elapsed() << std::endl;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}